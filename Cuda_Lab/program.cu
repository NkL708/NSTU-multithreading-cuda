#include "hip/hip_runtime.h"
﻿#include <ctime>
#include <iostream>

#include <stdio.h>
//#include <omp.h>
#include "hip/hip_runtime.h"
#include ""

__host__ __device__ bool isPalindrome(int num) 
{
	// Get number lenght
	int length = 0;
	for (int i = num; i > 0; i /= 10) {
		length++;
	}
	int* digits = new int[length];
	// Fill digits array
	int tempNum = num;
	for (int i = 0; i < length; i++) {
		digits[i] = tempNum % 10;
		tempNum /= 10;
	}
	// Comparison mirrored elements
	for (int begin = 0, end = length - 1; begin != end; begin++, end--) {
		if (digits[begin] != digits[end]) {
			return false;
		}
	}
	return true;
}

int getMaxPalindrome(int n)
{
	int maxPalindrome = 0;
	for (int num = 0; num < n; num++) {
		for (int firstPower = 0; pow(firstPower, 2) <= num; firstPower++) {
			int sum = 0;
			for (int numberOfSeq = firstPower; sum < num; numberOfSeq++) {
				sum += (int) pow(numberOfSeq, 2);
			}
			if (sum == num && isPalindrome(num)) {
				maxPalindrome = num;
			}
		}
	}
	return maxPalindrome;
}

void getSequence(int num)
{
	int lastPower = 0;
	for (int firstPower = 0; pow(firstPower, 2) <= num; firstPower++) {
		int sum = 0;;
		for (int numberOfSeq = firstPower; sum < num; numberOfSeq++) {
			lastPower = numberOfSeq;
			sum += (int)pow(numberOfSeq, 2);
		}
		if (sum == num && isPalindrome(num)) {
			break;
		}
	}
	for (int numberOfSeq = lastPower, sum = 0; sum < num; numberOfSeq--) {
		sum += (int)pow(numberOfSeq, 2);
		std::cout << numberOfSeq << "^2 + ";
	}
	std::cout << " = " << num << "\n\n";
}

int getCores(hipDeviceProp_t devProp)
{
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major) {
	case 2: // Fermi
		if (devProp.minor == 1) cores = mp * 48;
		else cores = mp * 32;
		break;
	case 3: // Kepler
		cores = mp * 192;
		break;
	case 5: // Maxwell
		cores = mp * 128;
		break;
	case 6: // Pascal
		if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
		else if (devProp.minor == 0) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	case 7: // Volta and Turing
		if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
		else printf("Unknown device type\n");
		break;
	case 8: // Ampere
		if (devProp.minor == 0) cores = mp * 64;
		else if (devProp.minor == 6) cores = mp * 128;
		else printf("Unknown device type\n");
		break;
	default:
		printf("Unknown device type\n");
		break;
	}
	return cores;
}

__global__ void getMaxPalindromeCUDA(int n, int result)
{
	for (int num = threadIdx.x; num < n; num += threadIdx.x) {
		for (int firstPower = 0; firstPower * firstPower <= num; firstPower++) {
			int sum = 0;
			for (int numberOfSeq = firstPower; sum < num; numberOfSeq++) {
				sum += numberOfSeq * numberOfSeq;
			}
			if (sum == num && isPalindrome(num)) {
				result = num;
			}
		}
	}
}

int main(int argc, char* argv[]) {
	double durationL, durationP;
	clock_t timeBegin, timeEnd;
	int num, result, cores;
	// Get CUDA cores
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	cores = getCores(device);
	if (argc > 1) {
		num = atoi(argv[1]);
	}
	else {
		num = 100000;
	}
	timeBegin = clock();
	result = getMaxPalindrome(num);
	timeEnd = clock();
	std::cout << "Linear result: " << result << std::endl;
	durationL = (double)(timeEnd - timeBegin) / CLOCKS_PER_SEC;
	timeBegin = clock();
	getMaxPalindromeCUDA << <1, cores >> > (num, result);
	timeEnd = clock();
	std::cout << "Parallel result: " << result << std::endl;
	durationP = (double)(timeEnd - timeBegin) / CLOCKS_PER_SEC;
	std::cout << "Linear time: " << durationL << std::endl;
	std::cout << "Parallel time: " << durationP << std::endl;
	std::cout << "Parallel faster than Linear on: " << durationL - durationP << std::endl;
	getSequence(result);
	return 0;
}